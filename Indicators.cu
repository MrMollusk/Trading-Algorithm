#include "hip/hip_runtime.h"
//CUDA lib streams
#include <hip/hip_runtime.h>
#include ""

#include <vector>
#include "indicatorsHEADER.h"
#include "API.h"
#include <math.h>

std::vector<float> jsonNumberExtractor(const std::vector<float>& vector, const int size) {
    std::vector<float> changedVector(vector.begin(), vector.begin() + std::min(size, (int)vector.size()));
    return changedVector;
}

float SMAKernelLaunch(const std::vector<float>& h_vector, int vectorElementNumber) {
    float* d_vector, * d_sum;
    float h_sum = 0;

    int vectorSize = sizeof(float) * vectorElementNumber;
    int threadsPerBlock = 32;
    int blocks = (vectorElementNumber + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc((void**)&d_vector, vectorSize);
    hipMalloc((void**)&d_sum, sizeof(float));

    hipMemset(d_sum, 0, sizeof(float));

    hipMemcpy(d_vector, h_vector.data(), vectorSize, hipMemcpyHostToDevice);

    closeSum << <blocks, threadsPerBlock >> > (d_vector, d_sum, vectorElementNumber);
    hipDeviceSynchronize();

    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vector);
    hipFree(d_sum);

    return h_sum / vectorElementNumber;
}

float EMAKernelLaunch(const std::vector<float>& h_vector, int ema) {
    const size_t vectorElementNumber = h_vector.size();
    float* d_vector, * d_ema;
    float smoothingFactor = 2.0 / (ema + 1);
    float initialEMA = 0.0;

    size_t vectorSize = sizeof(float) * vectorElementNumber;

    hipMalloc((void**)&d_vector, vectorSize);
    hipMalloc((void**)&d_ema, vectorSize);

    hipMemcpy(d_vector, h_vector.data(), vectorSize, hipMemcpyHostToDevice);

    for (int i = 0; i < ema; ++i) {
        initialEMA += h_vector[i];
    }
    initialEMA /= ema;

    hipMemcpy(&d_ema[ema - 1], &initialEMA, sizeof(float), hipMemcpyHostToDevice);

    EMA80Calculation<<<1, 1 >>> (d_vector, d_ema, smoothingFactor, ema, vectorElementNumber);
    hipDeviceSynchronize();

    std::vector<float> h_ema(vectorElementNumber);
    hipMemcpy(h_ema.data(), d_ema, vectorSize, hipMemcpyDeviceToHost);

    hipFree(d_vector);
    hipFree(d_ema);

    return h_ema[vectorElementNumber - 1];
}


std::vector<float> stochasticCalculation(float* h_high, float* h_low, float* h_close, int stockNumber, int timeStep, int backPeriod, int SMAPeriod) {
    float* d_high, * d_low, * d_close, * d_K, * d_D;

    hipMalloc((void**)&d_high, stockNumber * timeStep * sizeof(float));
    hipMalloc((void**)&d_low, stockNumber * timeStep * sizeof(float));
    hipMalloc((void**)&d_close, stockNumber * timeStep * sizeof(float));
    hipMalloc((void**)&d_K, stockNumber * timeStep * sizeof(float));
    hipMalloc((void**)&d_D, stockNumber * timeStep * sizeof(float));

    hipMemcpy(d_high, h_high, stockNumber * timeStep * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_low, h_low, stockNumber * timeStep * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_close, h_close, stockNumber * timeStep * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (stockNumber + blockSize - 1) / blockSize;
    size_t sharedMemSize = 2 * backPeriod * sizeof(float);

    stochasticKernel <<<gridSize, blockSize, sharedMemSize >>> (d_high, d_low, d_close, d_K, d_D, backPeriod, SMAPeriod, stockNumber, timeStep);
    hipDeviceSynchronize();

    std::vector<float> h_D(stockNumber * timeStep);
    hipMemcpy(h_D.data(), d_D, stockNumber * timeStep * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_high);
    hipFree(d_low);
    hipFree(d_close);
    hipFree(d_K);
    hipFree(d_D);

    return h_D;
}

//Kernel for sum of close values
__global__ void closeSum(float* d_vector, float* d_sum, const int size) {
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if (idx < size) {
        atomicAdd(d_sum, d_vector[idx]);
    }
}

//Kernel to calculate 80EMA
__global__ void EMA80Calculation(float* d_vector, float* d_ema, float smoothingFactor, int ema, const size_t size) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (size_t i = ema; i < size; ++i) {
            d_ema[i] = (d_vector[i] - d_ema[i - 1]) * smoothingFactor + d_ema[i - 1];
        }
    }
}

//Kernel for Stochastic Oscillator
__global__ void stochasticKernel(float* d_high, float* d_low, float* d_close, float* d_K, float* d_D, int backPeriod, int SMAPeriod, int stockNumber, int timeStep) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= stockNumber) return;

    extern __shared__ float sharedData[];
    float* sharedHighs = sharedData;
    float* sharedLows = &sharedHighs[backPeriod];

    for (int t = 0; t < timeStep; ++t) {
        if (threadIdx.x < backPeriod && t >= threadIdx.x) {
            sharedHighs[threadIdx.x] = d_high[idx * timeStep + t - threadIdx.x];
            sharedLows[threadIdx.x] = d_low[idx * timeStep + t - threadIdx.x];
        }
        else if (threadIdx.x < backPeriod) {
            sharedHighs[threadIdx.x] = 0.0;
            sharedLows[threadIdx.x] = 0.0;
        }
        __syncthreads();

        float highHigh = sharedHighs[0];
        float lowLow = sharedLows[0];
        for (int i = 1; i < backPeriod; ++i) {
            highHigh = fmax(highHigh, sharedHighs[i]);
            lowLow = fmin(lowLow, sharedLows[i]);
        }

        float currentClose = d_close[idx * timeStep + t];
        if (highHigh != lowLow) {
            d_K[idx * timeStep + t] = ((currentClose - lowLow) / (highHigh - lowLow)) * 100.0;
        }
        else {
            d_K[idx * timeStep + t] = 0.0;
        }

        __syncthreads();

        if (t >= SMAPeriod - 1) {
            float sumK = 0.0;
            for (int i = 0; i < SMAPeriod; ++i) {
                sumK += d_K[idx * timeStep + t - i];
            }
            d_D[idx * timeStep + t] = sumK / SMAPeriod;
        }
    }
}
